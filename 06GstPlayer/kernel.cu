#include <hip/hip_runtime.h>


#define NUM_THREAD_X 32
#define NUM_THREAD_Y 8

static int DivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__global__ void cuConvert_Rgb10A2_Rgba8(unsigned int *pSrc, unsigned char *pDst, int iWidth, int iHeight, int iSrcStride, int iDstStride)
{
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;
    if (dx >= iWidth || dy >= iHeight) {
        return;
    }

    int idxSrc = dy * iSrcStride + dx;
    int idxDst = dy * iDstStride + dx * 4;

    uint r = pSrc[idxSrc] >> 22;
    uint g = pSrc[idxSrc] >> 12 & 1023;
    uint b = pSrc[idxSrc] >> 2 & 1023;

    b = b >> 2;
    g = g >> 2;
    r = r >> 2;

    //pDst[idxDst] = r << 24 | g << 16 | b << 8 | 255;
    //pDst[idxDst] = b << 24 | g << 16 | r << 8 | 255;
    pDst[idxDst]     = r;
    pDst[idxDst + 1] = g;
    pDst[idxDst + 2] = b;
    pDst[idxDst + 3] = 255;
}

extern "C"
void Convert_Rgb10A2_Rgba8(unsigned int *pSrc, unsigned char *pDst, int iWidth, int iHeight, int iSrcStride, int iDstStride)
{
    dim3 numThreads(NUM_THREAD_X, NUM_THREAD_Y, 1);
    dim3 numBlocks(DivUp(iWidth, numThreads.x), DivUp(iHeight, numThreads.y));
    cuConvert_Rgb10A2_Rgba8<<<numBlocks, numThreads>>>(pSrc, pDst, iWidth, iHeight, iSrcStride, iDstStride);
}

__global__ void cuRgba8_ResizeHeight(unsigned char *pSrc, unsigned char *pDst, int iWidth, int iHeight, int iStride, bool expand)
{
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;

    if (dx >= iWidth || dy >= iHeight) {
        return;
    }

    int idxSrc = dy * iWidth + dx;
    int idxDst = dy * iWidth * 2 + dx;


    pDst[idxDst]          = pSrc[idxSrc];
    pDst[idxDst + iWidth] = pSrc[idxSrc];



//    int stepY = dy * iStride;
//    if (expand) {
//        stepY *= 2;
//    }
//    else {
//        stepY /= 2;
//    }

//    int idxSrc = dy * iStride + dx * 4;
//    int idxDst = stepY + dx * 4;

//    pDst[idxDst]     = pSrc[idxSrc];
//    pDst[idxDst + 1] = pSrc[idxSrc + 1];
//    pDst[idxDst + 2] = pSrc[idxSrc + 2];
//    pDst[idxDst + 3] = 255;
}

extern "C"
void rgba8_ResizeHeight(unsigned char *pSrc, unsigned char *pDst, int iWidth, int iHeight, int istride, bool expand)
{
    dim3 numThreads(NUM_THREAD_X, NUM_THREAD_Y, 1);
    dim3 numBlocks(DivUp(iWidth, numThreads.x), DivUp(iHeight, numThreads.y));

    cuRgba8_ResizeHeight<<<numBlocks, numThreads>>>(pSrc, pDst, iWidth, iHeight, istride, expand);
}



__global__ void cuRgba8_TopDownPack(unsigned char *pSrc1, unsigned char *pSrc2, unsigned char *pDst, int iWidth, int iHeight, int iStride)
{
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;

    if (dx >= iWidth || dy >= iHeight) {
        return;
    }

    int idxSrc = dy * iStride + dx * 4;
    int idxTopDst = dy / 2 * iStride + dx * 4;
    int idxdownDst = (dy / 2 + iHeight / 2) * iStride + dx * 4;

    pDst[idxTopDst]     = pSrc1[idxSrc];
    pDst[idxTopDst + 1] = pSrc1[idxSrc + 1];
    pDst[idxTopDst + 2] = pSrc1[idxSrc + 2];
    pDst[idxTopDst + 3] = 255;

    pDst[idxdownDst]     = pSrc2[idxSrc];
    pDst[idxdownDst + 1] = pSrc2[idxSrc + 1];
    pDst[idxdownDst + 2] = pSrc2[idxSrc + 2];
    pDst[idxdownDst + 3] = 255;
}

extern "C"
void rgba8_TopDownPack(unsigned char *pSrc1, unsigned char *pSrc2, unsigned char *pDst, int iWidth, int iHeight, int istride)
{
    dim3 numThreads(NUM_THREAD_X, NUM_THREAD_Y, 1);
    dim3 numBlocks(DivUp(iWidth, numThreads.x), DivUp(iHeight, numThreads.y));

    cuRgba8_TopDownPack<<<numBlocks, numThreads>>>(pSrc1, pSrc2, pDst, iWidth, iHeight, istride);
}


__global__ void cuRgba8_LeftRightPack(unsigned char *pSrc1, unsigned char *pSrc2, unsigned char *pDst, int iWidth, int iHeight, int iStride)
{
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;

    if (dx >= iWidth || dy >= iHeight) {
        return;
    }

        int idxSrc = dy * iStride + dx * 4;
        int idxLeftDst = dy * iStride + dx / 2 * 4;
        int idxRightDst = dy * iStride + (iWidth / 2 + dx / 2) * 4;

        pDst[idxLeftDst]     = pSrc1[idxSrc];
        pDst[idxLeftDst + 1] = pSrc1[idxSrc + 1];
        pDst[idxLeftDst + 2] = pSrc1[idxSrc + 2];
        pDst[idxLeftDst + 3] = 255;

        pDst[idxRightDst]     = pSrc2[idxSrc];
        pDst[idxRightDst + 1] = pSrc2[idxSrc + 1];
        pDst[idxRightDst + 2] = pSrc2[idxSrc + 2];
        pDst[idxRightDst + 3] = 255;
}

extern "C"
void rgba8_LeftRightPack(unsigned char *pSrc1, unsigned char *pSrc2, unsigned char *pDst, int iWidth, int iHeight, int istride)
{
    dim3 numThreads(NUM_THREAD_X, NUM_THREAD_Y, 1);
    dim3 numBlocks(DivUp(iWidth, numThreads.x), DivUp(iHeight, numThreads.y));

    cuRgba8_LeftRightPack<<<numBlocks, numThreads>>>(pSrc1, pSrc2, pDst, iWidth, iHeight, istride);
}


__global__ void cuRgba8_Blending(unsigned char *pSrc, unsigned char *pTopImageLayer, unsigned char *pDst, int iWidth, int iHeight, int iStride)
{
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;

    if (dx >= iWidth || dy >= iHeight) {
        return;
    }

    int idx = dy * iStride + dx * 4;

    uint r = 0;
    uint g = 0;
    uint b = 0;

    if((pTopImageLayer[idx+3] == 0) ||
        (pTopImageLayer[idx] < 30 && pTopImageLayer[idx+1] < 30
         && pTopImageLayer[idx+2] < 30) ) {

        r = pSrc[idx];
        g = pSrc[idx + 1];
        b = pSrc[idx + 2];
    } else {
        r = pTopImageLayer[idx];
        g = pTopImageLayer[idx + 1];
        b = pTopImageLayer[idx + 2];
    }

    pDst[idx]     = r;
    pDst[idx + 1] = g;
    pDst[idx + 2] = b;
    pDst[idx + 3] = 255;
}

extern "C"
void rgba8_Blending(unsigned char *pSrc, unsigned char *pTopImageLayer, unsigned char *pDst, int iWidth, int iHeight, int istride)
{
    dim3 numThreads(NUM_THREAD_X, NUM_THREAD_Y, 1);
    dim3 numBlocks(DivUp(iWidth, numThreads.x), DivUp(iHeight, numThreads.y));

    cuRgba8_Blending<<<numBlocks, numThreads>>>(pSrc, pTopImageLayer, pDst, iWidth, iHeight, istride);
}


__global__ void cuYUYV422_to_rgba8(const unsigned char *yuvImage, unsigned char *rgbaImage, int width, int height)
{
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;

    // 2 pixels per process unit
    if (dx < width/2 && dy < height) {
        int index = dy * width + dx * 2;
        int yuyvIdx = index * 2;
        int rgbaIdx = index * 4;

        // fetch
        unsigned char y0 = yuvImage[yuyvIdx];
        unsigned char u  = yuvImage[yuyvIdx + 1];
        unsigned char y1 = yuvImage[yuyvIdx + 2];
        unsigned char v  = yuvImage[yuyvIdx + 3];

        // YUYV to RGB
        int r0 = y0 + 1.402 * (v - 128);
        int g0 = y0 - 0.344136 * (u - 128) - 0.714136 * (v - 128);
        int b0 = y0 + 1.772 * (u - 128);

        int r1 = y1 + 1.402 * (v - 128);
        int g1 = y1 - 0.344136 * (u - 128) - 0.714136 * (v - 128);
        int b1 = y1 + 1.772 * (u - 128);

        // clamp
        r0 = min(max(r0, 0), 255);
        g0 = min(max(g0, 0), 255);
        b0 = min(max(b0, 0), 255);

        r1 = min(max(r1, 0), 255);
        g1 = min(max(g1, 0), 255);
        b1 = min(max(b1, 0), 255);

        // assign
        rgbaImage[rgbaIdx    ] = r0;
        rgbaImage[rgbaIdx + 1] = g0;
        rgbaImage[rgbaIdx + 2] = b0;
        rgbaImage[rgbaIdx + 3] = 255;

        rgbaImage[rgbaIdx + 4] = r1;
        rgbaImage[rgbaIdx + 5] = g1;
        rgbaImage[rgbaIdx + 6] = b1;
        rgbaImage[rgbaIdx + 7] = 255;
    }
}

extern "C"
void YUYV422_to_rgba8(const unsigned char *yuvImage, unsigned char *rgbaImage, int iWidth, int iHeight)
{
    dim3 blockSize(NUM_THREAD_X, NUM_THREAD_Y);
    dim3 gridSize(DivUp(iWidth, blockSize.x), DivUp(iHeight, blockSize.y));

    cuYUYV422_to_rgba8<<<gridSize, blockSize>>>(yuvImage, rgbaImage, iWidth, iHeight);
}

__global__ void cuYUYV422_to_rgba10A2(const unsigned char *yuvImage, unsigned int *rgbaImage, int width, int height)
{
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;

    // 2 pixels per process unit
    if (dx < width/2 && dy < height) {
        int index = dy * width + dx * 2;
        int yuyvIdx = index * 2;
        int rgbaIdx = index;

        // fetch
        unsigned char y0 = yuvImage[yuyvIdx];
        unsigned char u  = yuvImage[yuyvIdx + 1];
        unsigned char y1 = yuvImage[yuyvIdx + 2];
        unsigned char v  = yuvImage[yuyvIdx + 3];

        // YUYV to RGB
        int r0 = y0 + 1.402 * (v - 128);
        int g0 = y0 - 0.344136 * (u - 128) - 0.714136 * (v - 128);
        int b0 = y0 + 1.772 * (u - 128);

        int r1 = y1 + 1.402 * (v - 128);
        int g1 = y1 - 0.344136 * (u - 128) - 0.714136 * (v - 128);
        int b1 = y1 + 1.772 * (u - 128);

        // clamp
        unsigned int ur0 = min(max(r0, 0), 255);
        unsigned int ug0 = min(max(g0, 0), 255);
        unsigned int ub0 = min(max(b0, 0), 255);

        unsigned int ur1 = min(max(r1, 0), 255);
        unsigned int ug1 = min(max(g1, 0), 255);
        unsigned int ub1 = min(max(b1, 0), 255);

        // assign
        rgbaImage[rgbaIdx    ] = ur0 << 24 | ug0 << 14 | ub0 << 4 | 3;
        rgbaImage[rgbaIdx + 1] = ur1 << 24 | ug1 << 14 | ub1 << 4 | 3;
    }
}

extern "C"
void YUYV422_to_rgba10A2(const unsigned char *yuvImage, unsigned int *rgbaImage, int iWidth, int iHeight)
{
    dim3 blockSize(NUM_THREAD_X, NUM_THREAD_Y);
    dim3 gridSize(DivUp(iWidth, blockSize.x), DivUp(iHeight, blockSize.y));

    cuYUYV422_to_rgba10A2<<<gridSize, blockSize>>>(yuvImage, rgbaImage, iWidth, iHeight);
}


__global__ void cuNV12_21_to_rgba8(const unsigned char *pSrc, int uvIndex, unsigned char *pDst, int iWidth, int iHeight, bool isNv12)
{
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;

    if (dx < iWidth && dy < iHeight) {

        unsigned char y, u, v;
        int idxY, idxU, idxV;

        idxY = dy * iWidth + dx;
        y = pSrc[idxY];

        int idx1, idx2;
        int uvStep =  dy / 2 * iWidth + dx;

        if (dx % 2) {
            //column odd
            idx1 = uvStep - 1;
            idx2 = uvStep;
        }
        else {
            //column even
            idx1 = uvStep;
            idx2 = uvStep + 1;
        }

        idxU = isNv12 ? idx1 : idx2;
        idxV = isNv12 ? idx2 : idx1;

        u = pSrc[uvIndex + idxU];
        v = pSrc[uvIndex + idxV];

        int r = y + 1.402 * (v - 128);
        int g = y - 0.344136 * (u - 128) - 0.714136 * (v - 128);
        int b = y + 1.772 * (u - 128);

        r = min(max(r, 0), 255);
        g = min(max(g, 0), 255);
        b = min(max(b, 0), 255);

        int idxRgba = (dy * iWidth + dx) * 4;
        pDst[idxRgba] = r;
        pDst[idxRgba + 1] = g;
        pDst[idxRgba + 2] = b;
        pDst[idxRgba + 3] = 255;
    }
}

extern "C"
void nv12_21_to_rgba8(const unsigned char *pSrc, int uvIndex, unsigned char *pDst, int iWidth, int iHeight, bool isNv12)
{
    dim3 blockSize(NUM_THREAD_X, NUM_THREAD_Y);
    dim3 gridSize(DivUp(iWidth, blockSize.x), DivUp(iHeight, blockSize.y));

    cuNV12_21_to_rgba8<<<gridSize, blockSize>>>(pSrc, uvIndex, pDst, iWidth, iHeight, isNv12);
}

__global__ void cuI420_YV12_to_rgba8(const unsigned char *pSrc, unsigned char *pDst, int iWidth, int iHeight, bool isI420)
{
    const int dx = blockIdx.x * blockDim.x + threadIdx.x;
    const int dy = blockIdx.y * blockDim.y + threadIdx.y;

    if (dx < iWidth && dy < iHeight) {
        unsigned char y, u, v;

        int indexY = dy * iWidth + dx;
        y = pSrc[indexY];

        int idx1 = iWidth * iHeight;
        int idx2 = iWidth * iHeight * 5 / 4;

        int idxU = isI420 ? idx1 : idx2;
        int idxV = isI420 ? idx2 : idx1;

        /* core */
        int stepUV = (dy / 2 * iWidth + dx) / 2;

        u = pSrc[stepUV + idxU];
        v = pSrc[stepUV + idxV];

        int r = y + 1.402 * (v - 128);
        int g = y - 0.344136 * (u - 128) - 0.714136 * (v - 128);
        int b = y + 1.772 * (u - 128);

        r = min(max(r, 0), 255);
        g = min(max(g, 0), 255);
        b = min(max(b, 0), 255);

        int idxRgba = (dy * iWidth + dx) * 4;
        pDst[idxRgba] = r;
        pDst[idxRgba + 1] = g;
        pDst[idxRgba + 2] = b;
        pDst[idxRgba + 3] = 255;
    }
}

extern "C"
void i420_yv12_to_rgba8(const unsigned char *pSrc, unsigned char *pDst, int iWidth, int iHeight, bool isI420)
{
    dim3 blockSize(NUM_THREAD_X, NUM_THREAD_Y);
    dim3 gridSize(DivUp(iWidth, blockSize.x), DivUp(iHeight, blockSize.y));

    cuI420_YV12_to_rgba8<<<gridSize, blockSize>>>(pSrc, pDst, iWidth, iHeight, isI420);
}

